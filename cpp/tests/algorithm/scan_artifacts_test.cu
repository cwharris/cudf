#include <algorithm>

#include <cudf_test/base_fixture.hpp>
#include <cudf_test/cudf_gtest.hpp>

#include <cudf/algorithm/csv_gpu_row_count.cuh>
#include <cudf/algorithm/scan_artifacts.cuh>
#include <cudf/utilities/span.hpp>
#include "rmm/device_buffer.hpp"
#include "rmm/device_scalar.hpp"
#include "rmm/device_uvector.hpp"

#include <rmm/thrust_rmm_allocator.h>

#include <thrust/iterator/constant_iterator.h>

class InclusiveCopyIfTest : public cudf::test::BaseFixture {
};

struct simple_output {
  fsm_output<uint32_t> a;
  fsm_output<double> b;

  inline __device__ simple_output operator+(simple_output other) const
  {
    return {
      a + other.a,
      b + other.b,
    };
  }
};

struct simple_state {
  uint32_t sum;
  inline __device__ simple_state operator+(simple_state other) const
  {
    return {
      sum + other.sum,
    };
  }
};

struct simple_state_seed_op {
  inline __device__ simple_state operator()(uint32_t idx, uint32_t input)  //
  {
    return {};
  }
};

struct simple_state_step_op {
  template <bool output_enabled>
  inline __device__ simple_state operator()(  //
    simple_output& outputs,
    simple_state prev_state,
    uint32_t rhs)
  {
    auto state = simple_state{
      prev_state.sum + rhs,
    };

    if (state.sum % 3 == 0) { outputs.a.emit<output_enabled>(state.sum); }
    if (state.sum % 2 == 0) { outputs.b.emit<output_enabled>(state.sum * 2.0); }

    return state;
  }
};

struct simple_state_join_op {
  inline __device__ simple_state operator()(simple_state lhs, simple_state rhs)  //
  {
    return lhs + rhs;
  }
};

TEST_F(InclusiveCopyIfTest, CanScanSelectIf)
{
  auto input = thrust::make_constant_iterator<uint32_t>(1);

  auto seed_op = simple_state_seed_op{};
  auto step_op = simple_state_step_op{};
  auto join_op = simple_state_join_op{};

  const uint32_t input_size             = (1 << 15) + 4;
  const uint32_t expected_output_size_a = input_size / 3;
  const uint32_t expected_output_size_b = input_size / 2;

  thrust::device_vector<uint32_t> d_input(input, input + input_size);

  auto d_output_state = rmm::device_scalar<simple_state>();
  auto d_output       = rmm::device_scalar<simple_output>();

  rmm::device_buffer temp_storage;

  // phase 1: count outputs.
  temp_storage = scan_artifacts(std::move(temp_storage),  //
                                d_input.begin(),
                                d_input.end(),
                                d_output_state.data(),
                                d_output.data(),
                                seed_op,
                                step_op,
                                join_op);

  auto h_output       = d_output.value();
  auto h_output_state = d_output_state.value();

  EXPECT_EQ(input_size, h_output_state.sum);
  EXPECT_EQ(input_size, h_output_state.sum);

  ASSERT_EQ(expected_output_size_a, h_output.a.output_count);
  ASSERT_EQ(expected_output_size_b, h_output.b.output_count);

  // phase 2: allocate outputs

  auto output_a = rmm::device_uvector<uint32_t>(h_output.a.output_count, 0);
  auto output_b = rmm::device_uvector<double>(h_output.b.output_count, 0);

  h_output                 = {};
  h_output.a.output_buffer = output_a.data();
  h_output.b.output_buffer = output_b.data();

  d_output.set_value(h_output);
  d_output_state.set_value({});

  temp_storage = scan_artifacts(std::move(temp_storage),  //
                                d_input.begin(),
                                d_input.end(),
                                d_output_state.data(),
                                d_output.data(),
                                seed_op,
                                step_op,
                                join_op);

  h_output       = d_output.value();
  h_output_state = d_output_state.value();

  EXPECT_EQ(input_size, h_output_state.sum);
  EXPECT_EQ(input_size, h_output_state.sum);

  ASSERT_EQ(expected_output_size_a, h_output.a.output_count);
  ASSERT_EQ(expected_output_size_b, h_output.b.output_count);

  ASSERT_EQ(output_a.data(), h_output.a.output_buffer);
  ASSERT_EQ(output_b.data(), h_output.b.output_buffer);

  auto h_output_a = std::vector<uint32_t>(h_output.a.output_count);
  auto h_output_b = std::vector<double>(h_output.b.output_count);

  hipMemcpy(h_output_a.data(),
             h_output.a.output_buffer,
             h_output.a.output_count * sizeof(uint32_t),
             hipMemcpyDeviceToHost);

  hipMemcpy(h_output_b.data(),
             h_output.b.output_buffer,
             h_output.b.output_count * sizeof(double),
             hipMemcpyDeviceToHost);

  for (uint32_t i = 0; i < h_output_a.size(); i++) {
    ASSERT_EQ(static_cast<uint32_t>(i * 3 + 3), h_output_a[i]);
  }

  for (uint32_t i = 0; i < h_output_b.size(); i++) {
    ASSERT_EQ(static_cast<double>(i * 4.0 + 4), h_output_b[i]);
  }
}

TEST_F(InclusiveCopyIfTest, CanTransitionCsvStates)
{
  auto input = std::string(
    "hello, world\n"
    "and,\"not\nh,ing\"\n"
    "new\n"
    "hello, world\n"
    "and,\"not\nh,ing\"\n"
    "new\n"
    "hello, world\n"
    "and,\"not\nh,ing\"\n"
    "new\n");

  auto d_input = rmm::device_vector<char>(input.c_str(), input.c_str() + input.size());

  auto d_row_offsets = cudf::io::detail::csv_gather_row_offsets(d_input);

  ASSERT_EQ(static_cast<uint32_t>(9), d_row_offsets.size());

  auto h_row_offsets = std::vector<uint32_t>(d_row_offsets.size());

  hipStreamSynchronize(0);

  hipMemcpy(h_row_offsets.data(),  //
             d_row_offsets.data(),
             d_row_offsets.size() * sizeof(uint32_t),
             hipMemcpyDeviceToHost);

  EXPECT_EQ(static_cast<uint32_t>(0), h_row_offsets[0]);
  EXPECT_EQ(static_cast<uint32_t>(13), h_row_offsets[1]);
  EXPECT_EQ(static_cast<uint32_t>(29), h_row_offsets[2]);

  EXPECT_EQ(static_cast<uint32_t>(33), h_row_offsets[3]);
  EXPECT_EQ(static_cast<uint32_t>(46), h_row_offsets[4]);
  EXPECT_EQ(static_cast<uint32_t>(62), h_row_offsets[5]);

  EXPECT_EQ(static_cast<uint32_t>(66), h_row_offsets[6]);
  EXPECT_EQ(static_cast<uint32_t>(79), h_row_offsets[7]);
  EXPECT_EQ(static_cast<uint32_t>(95), h_row_offsets[8]);
}

CUDF_TEST_PROGRAM_MAIN()
