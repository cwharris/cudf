#include <algorithm>

#include <cudf_test/base_fixture.hpp>
#include <cudf_test/cudf_gtest.hpp>

#include <cudf/algorithm/csv_gpu_row_count.cuh>
#include <cudf/algorithm/scan_artifacts.cuh>
#include <cudf/utilities/span.hpp>
#include "gtest/gtest.h"
#include "rmm/device_buffer.hpp"
#include "rmm/device_scalar.hpp"
#include "rmm/device_uvector.hpp"

#include <rmm/thrust_rmm_allocator.h>

#include <thrust/iterator/constant_iterator.h>

class InclusiveCopyIfTest : public cudf::test::BaseFixture {
};

struct simple_output {
  fsm_output<uint32_t> a;
  fsm_output<double> b;

  inline __device__ simple_output operator+(simple_output other) const
  {
    // printf("bid(%i) tid(%i): simple_output (%i %i) + (%i %i)\n",  //
    //        blockIdx.x,
    //        threadIdx.x,
    //        a.output_count,
    //        b.output_count,
    //        other.a.output_count,
    //        other.b.output_count);

    return {
      a + other.a,
      b + other.b,
    };
  }
};

struct simple_state {
  uint32_t sum;
  inline __device__ simple_state operator+(simple_state other) const
  {
    // printf("bid(%i) tid(%i): simple_state %i + %i\n",  //
    //        blockIdx.x,
    //        threadIdx.x,
    //        sum,
    //        other.sum);

    return {
      sum + other.sum,
    };
  }
};

struct simple_state_seed_op {
  inline __device__ simple_state operator()(uint32_t idx, uint32_t input)  //
  {
    // printf("bid(%i) tid(%i): simple_state_seed_op %i + %i\n",  //
    //        blockIdx.x,
    //        threadIdx.x,
    //        idx,
    //        input);

    return {};
  }
};

struct simple_state_step_op {
  template <bool output_enabled>
  inline __device__ simple_state operator()(  //
    simple_output& outputs,
    simple_state prev_state,
    uint32_t rhs)
  {
    // printf("bid(%i) tid(%i): simple_state_step_op (%i, %i), %i, %i\n",  //
    //        blockIdx.x,
    //        threadIdx.x,
    //        outputs.a.output_count,
    //        outputs.b.output_count,
    //        prev_state.sum,
    //        rhs);

    auto state = simple_state{
      prev_state.sum + rhs,
    };

    if (state.sum % 3 == 0) { outputs.a.emit<output_enabled>(state.sum); }
    // if (state.sum % 2 == 0) { outputs.b.emit<output_enabled>(state.sum * 2.0); }

    return state;
  }
};

struct simple_state_join_op {
  inline __device__ simple_state operator()(simple_state lhs, simple_state rhs)  //
  {
    // printf("bid(%i) tid(%i): simple_state_step_op %i => %i\n",  //
    //        blockIdx.x,
    //        threadIdx.x,
    //        lhs.sum,
    //        rhs.sum);

    return lhs + rhs;
  }
};

TEST_F(InclusiveCopyIfTest, CanScanSelectIf)
{
  auto input = thrust::make_constant_iterator<uint32_t>(1);

  auto seed_op = simple_state_seed_op{};
  auto step_op = simple_state_step_op{};
  auto join_op = simple_state_join_op{};

  const uint32_t input_size             = 1 << 15;
  const uint32_t expected_output_size_a = input_size / 3;
  // const uint32_t expected_output_size_b = input_size / 2;

  thrust::device_vector<uint32_t> d_input(input, input + input_size);

  auto d_output_state = rmm::device_scalar<simple_state>();
  auto d_output       = rmm::device_scalar<simple_output>();

  rmm::device_buffer temp_storage;

  // phase 1: count outputs.
  temp_storage = scan_artifacts(std::move(temp_storage),  //
                                d_input.begin(),
                                d_input.end(),
                                d_output_state.data(),
                                d_output.data(),
                                seed_op,
                                step_op,
                                join_op);

  auto h_output       = d_output.value();
  auto h_output_state = d_output_state.value();

  EXPECT_EQ(input_size, h_output_state.sum);
  EXPECT_EQ(input_size, h_output_state.sum);

  ASSERT_EQ(expected_output_size_a, h_output.a.output_count);
  // ASSERT_EQ(expected_output_size_b, h_output.b.output_count);

  // phase 2: allocate outputs

  auto output_a = rmm::device_uvector<uint32_t>(h_output.a.output_count, 0);
  auto output_b = rmm::device_uvector<double>(h_output.b.output_count, 0);

  h_output                 = {};
  h_output.a.output_buffer = output_a.data();
  h_output.b.output_buffer = output_b.data();

  d_output.set_value(h_output);
  d_output_state.set_value({});

  temp_storage = scan_artifacts(std::move(temp_storage),  //
                                d_input.begin(),
                                d_input.end(),
                                d_output_state.data(),
                                d_output.data(),
                                seed_op,
                                step_op,
                                join_op);

  h_output       = d_output.value();
  h_output_state = d_output_state.value();

  EXPECT_EQ(input_size, h_output_state.sum);
  EXPECT_EQ(input_size, h_output_state.sum);

  ASSERT_EQ(expected_output_size_a, h_output.a.output_count);
  // ASSERT_EQ(expected_output_size_b, h_output.b.output_count);

  ASSERT_EQ(output_a.data(), h_output.a.output_buffer);
  // ASSERT_EQ(output_b.data(), h_output.b.output_buffer);

  auto h_output_a = std::vector<uint32_t>(h_output.a.output_count);
  // auto h_output_b = std::vector<double>(h_output.a.output_count);

  hipMemcpy(h_output_a.data(),
             h_output.a.output_buffer,
             h_output.a.output_count * sizeof(uint32_t),
             hipMemcpyDeviceToHost);

  // hipMemcpy(h_output_b.data(),
  //            h_output.b.output_buffer,
  //            h_output.b.output_count * sizeof(double),
  //            hipMemcpyDeviceToHost);

  for (uint32_t i = 0; i < h_output_a.size(); i++) {
    EXPECT_EQ(static_cast<uint32_t>(i * 3) + 3, h_output_a[i]);
    // EXPECT_EQ(static_cast<uint32_t>(-1), h_output_a[i]);
  }

  // for (uint32_t i = 0; i < h_output_b.size(); i++) {
  //   // EXPECT_EQ(static_cast<double>(i * 3), h_output_b[i]);
  //   ASSERT_EQ(static_cast<double>(-1), h_output_b[i]);
  // }
}

TEST_F(InclusiveCopyIfTest, CanTransitionCsvStates)
{
  // auto input = std::string("hello, world");

  // auto d_input = rmm::device_vector<char>(input.c_str(), input.c_str() + input.size());

  // auto d_row_offsets = csv_gather_row_offsets(d_input);

  // thrust::host_vector<uint32_t> h_row_offsets(d_row_offsets.size());

  // hipMemcpy(h_row_offsets.data(),  //
  //            d_row_offsets.data(),
  //            d_row_offsets.size() * sizeof(char),
  //            hipMemcpyDeviceToHost);

  // ASSERT_EQ(static_cast<uint32_t>(0), h_row_offsets.size());

  // auto d_result = scan_artifacts<uint32_t>(d_input.begin(),  //
  //                                          d_input.end(),
  //                                          seed_op,
  //                                          scan_op,
  //                                          intersect_op);

  // thrust::host_vector<uint32_t> h_result(d_result.size());
  // hipMemcpy(
  //   h_result.data(), d_result.data(), sizeof(uint32_t) * d_result.size(),
  //   hipMemcpyDeviceToHost);

  // for (uint32_t i = 0; i < h_result.size(); i++) {  //
  //   ASSERT_EQ(static_cast<uint32_t>((i / 2) * 3 + 3), h_result[i]);
  // }
}

CUDF_TEST_PROGRAM_MAIN()
