#include <algorithm>

#include <cudf_test/base_fixture.hpp>
#include <cudf_test/cudf_gtest.hpp>

#include <cudf/algorithm/csv_gpu_row_count.cuh>
#include <cudf/algorithm/scan_artifacts.cuh>
#include <cudf/utilities/span.hpp>
#include "gtest/gtest.h"
#include "rmm/device_scalar.hpp"
#include "rmm/device_uvector.hpp"

#include <rmm/thrust_rmm_allocator.h>

#include <thrust/iterator/constant_iterator.h>

class InclusiveCopyIfTest : public cudf::test::BaseFixture {
};

struct simple_outputs {
  fsm_output<uint32_t> a;
  fsm_output<double> b;

  inline __device__ simple_outputs operator+(simple_outputs other) const
  {
    // printf("bid(%i) tid(%i): simple_outputs (%i %i) + (%i %i)\n",  //
    //        blockIdx.x,
    //        threadIdx.x,
    //        a.output_count,
    //        b.output_count,
    //        other.a.output_count,
    //        other.b.output_count);

    return {
      a + other.a,
      b + other.b,
    };
  }
};

struct simple_state {
  uint32_t sum;
  inline __device__ simple_state operator+(simple_state other) const
  {
    // printf("bid(%i) tid(%i): simple_state %i + %i\n",  //
    //        blockIdx.x,
    //        threadIdx.x,
    //        sum,
    //        other.sum);

    return {
      sum + other.sum,
    };
  }
};

struct simple_state_seed_op {
  inline __device__ simple_state operator()(uint32_t idx, uint32_t input)  //
  {
    // printf("bid(%i) tid(%i): simple_state_seed_op %i + %i\n",  //
    //        blockIdx.x,
    //        threadIdx.x,
    //        idx,
    //        input);

    return {};
  }
};

struct simple_state_step_op {
  template <bool output_enabled>
  inline __device__ simple_state operator()(  //
    simple_outputs& outputs,
    simple_state prev_state,
    uint32_t rhs)
  {
    // printf("bid(%i) tid(%i): simple_state_step_op (%i, %i), %i, %i\n",  //
    //        blockIdx.x,
    //        threadIdx.x,
    //        outputs.a.output_count,
    //        outputs.b.output_count,
    //        prev_state.sum,
    //        rhs);

    auto state = simple_state{
      prev_state.sum + rhs,
    };

    if (prev_state.sum % 3 == 0) {
      outputs.a.emit<output_enabled>(state.sum);
      outputs.b.emit<output_enabled>(state.sum * 2.0);
    }

    return state;
  }
};

struct simple_state_join_op {
  inline __device__ simple_state operator()(simple_state lhs, simple_state rhs)  //
  {
    // printf("bid(%i) tid(%i): simple_state_step_op %i => %i\n",  //
    //        blockIdx.x,
    //        threadIdx.x,
    //        lhs.sum,
    //        rhs.sum);

    return lhs + rhs;
  }
};

TEST_F(InclusiveCopyIfTest, CanScanSelectIf)
{
  auto input = thrust::make_constant_iterator<uint32_t>(1);

  auto seed_op = simple_state_seed_op{};
  auto step_op = simple_state_step_op{};
  auto join_op = simple_state_join_op{};

  const uint32_t input_size = 1 << 10;

  thrust::device_vector<uint32_t> d_input(input, input + input_size);

  auto d_output_state = rmm::device_scalar<simple_state>();
  auto d_output       = rmm::device_scalar<simple_outputs>();

  rmm::device_buffer temp_storage;

  // phase 1: count outputs.
  temp_storage = scan_artifacts(std::move(temp_storage),  //
                                d_input.begin(),
                                d_input.end(),
                                d_output_state.data(),
                                d_output.data(),
                                seed_op,
                                step_op,
                                join_op);

  auto h_outputs = d_output.value();

  EXPECT_EQ(static_cast<uint32_t>(input_size), d_output_state.value().sum);
  EXPECT_EQ(static_cast<uint32_t>(input_size), d_output_state.value().sum);

  EXPECT_EQ(static_cast<uint32_t>(input_size / 3), h_outputs.a.output_count);
  EXPECT_EQ(static_cast<uint32_t>(input_size / 3), h_outputs.b.output_count);

  // phase 2: allocate outputs

  rmm::device_uvector<uint32_t> output_a(h_outputs.a.output_count, 0);
  rmm::device_uvector<double> output_b(h_outputs.a.output_count, 0);

  h_outputs.a.output_buffer = output_a.data();
  h_outputs.b.output_buffer = output_b.data();
}

TEST_F(InclusiveCopyIfTest, CanTransitionCsvStates)
{
  // auto input = std::string("hello, world");

  // auto d_input = rmm::device_vector<char>(input.c_str(), input.c_str() + input.size());

  // auto d_row_offsets = csv_gather_row_offsets(d_input);

  // thrust::host_vector<uint32_t> h_row_offsets(d_row_offsets.size());

  // hipMemcpy(h_row_offsets.data(),  //
  //            d_row_offsets.data(),
  //            d_row_offsets.size() * sizeof(char),
  //            hipMemcpyDeviceToHost);

  // ASSERT_EQ(static_cast<uint32_t>(0), h_row_offsets.size());

  // auto d_result = scan_artifacts<uint32_t>(d_input.begin(),  //
  //                                          d_input.end(),
  //                                          seed_op,
  //                                          scan_op,
  //                                          intersect_op);

  // thrust::host_vector<uint32_t> h_result(d_result.size());
  // hipMemcpy(
  //   h_result.data(), d_result.data(), sizeof(uint32_t) * d_result.size(),
  //   hipMemcpyDeviceToHost);

  // for (uint32_t i = 0; i < h_result.size(); i++) {  //
  //   ASSERT_EQ(static_cast<uint32_t>((i / 2) * 3 + 3), h_result[i]);
  // }
}

CUDF_TEST_PROGRAM_MAIN()
