#include <algorithm>
#include <cudf_test/base_fixture.hpp>
#include <cudf_test/cudf_gtest.hpp>

#include <cudf/algorithm/scan_copy_if.cuh>
#include <cudf/utilities/span.hpp>

#include <rmm/thrust_rmm_allocator.h>

#include <thrust/iterator/constant_iterator.h>

class InclusiveCopyIfTest : public cudf::test::BaseFixture {
};

struct simple_op {
  inline constexpr uint32_t operator()(uint32_t lhs, uint32_t rhs) { return lhs + rhs; }
  inline constexpr bool operator()(uint32_t value) { return value % 3 == 0; }
};

TEST_F(InclusiveCopyIfTest, CanScanSelectIf)
{
  auto input = thrust::make_constant_iterator<uint32_t>(1);

  auto op = simple_op{};

  // const uint32_t size = 1 << 24;
  const uint32_t input_size = 1 << 30;

  thrust::device_vector<uint32_t> h_input(input, input + input_size);

  auto d_result = scan_copy_if(h_input.begin(), h_input.end(), op, op);

  thrust::host_vector<uint32_t> h_result(d_result.size());

  hipMemcpy(
    h_result.data(), d_result.data(), sizeof(uint32_t) * d_result.size(), hipMemcpyDeviceToHost);

  // 4096 / 3 = 1365.333...
  ASSERT_EQ(static_cast<uint32_t>(input_size / 3), h_result.size());

  for (uint32_t i = 0; i < h_result.size(); i++) {  //
    ASSERT_EQ(static_cast<uint32_t>(i * 3 + 3), h_result[i]);
  }
}

struct successive_capitalization_state {
  char curr;
  char prev;
};

struct successive_capitalization_op {
  inline constexpr successive_capitalization_state operator()(  //
    successive_capitalization_state lhs,
    successive_capitalization_state rhs)
  {
    return {rhs.curr, lhs.curr};
  }

  static inline constexpr bool is_capital(char value) { return value >= 'A' and value <= 'Z'; }

  inline __device__ bool operator()(successive_capitalization_state value)
  {
    return is_capital(value.prev) and is_capital(value.curr);
  }
};

TEST_F(InclusiveCopyIfTest, CanDetectSuccessiveCapitals)
{
  auto input_str = std::string("AbcDeFGLiJKlMnoP");

  auto input = rmm::device_vector<successive_capitalization_state>(input_str.size());

  std::transform(input_str.begin(),  //
                 input_str.end(),
                 input.begin(),
                 [](char value) { return successive_capitalization_state{value}; });

  auto op = successive_capitalization_op{};

  auto d_result = scan_copy_if(  //
    input.begin(),
    input.end(),
    op,
    op);

  auto h_result = thrust::host_vector<successive_capitalization_state>(d_result.size());

  hipMemcpy(h_result.data(),  //
             d_result.data(),
             d_result.size() * sizeof(successive_capitalization_state),
             hipMemcpyDeviceToHost);

  ASSERT_EQ(static_cast<uint32_t>(3), h_result.size());

  EXPECT_EQ(static_cast<char>('G'), h_result[0].curr);
  EXPECT_EQ(static_cast<char>('L'), h_result[1].curr);
  EXPECT_EQ(static_cast<char>('K'), h_result[2].curr);
}

enum class csv_token { unknown, comment_start, comment, new_record };

enum class csv_state { nominal, commented };

struct csv_token_parse_state {
  char c;
  csv_token token;
  csv_state state;
};

csv_token_parse_state operator+(  //
  csv_token_parse_state const& lhs,
  csv_token_parse_state const& rhs)
{
  csv_token_parse_state result;

  result.c = rhs.c;

  switch (lhs.state) {
    case csv_state::nominal: {
      if (lhs.c == '\n') {
        if (rhs.c == '#') {
          return {rhs.c, csv_token::comment_start, csv_state::commented};
        } else {
          return {rhs.c, csv_token::new_record, csv_state::nominal};
        }
      }
      return {rhs.c, csv_token::unknown, csv_state::nominal};
    }
    case csv_state::commented: {
      if (lhs.c == '\n') {
        if (rhs.c == '#') {
          return {rhs.c, csv_token::comment_start, csv_state::commented};
        } else {
          return {rhs.c, csv_token::new_record, csv_state::nominal};
        }
      }
      return {rhs.c, csv_token::comment, csv_state::commented};
    }
  }

  return result;
}

struct csv_token_parse_op {
  inline __device__ csv_token_parse_state operator()(  //
    csv_token_parse_state lhs,
    csv_token_parse_state rhs)
  {
    return rhs;
  }
  inline __device__ bool operator()(csv_token_parse_state value) { return true; }
};

TEST_F(InclusiveCopyIfTest, CanParseCsv)
{
  auto input_str = std::string(
    "hello, world\n"
    "new, record\n");

  auto input = rmm::device_vector<successive_capitalization_state>(input_str.size());

  std::transform(input_str.begin(),  //
                 input_str.end(),
                 input.begin(),
                 [](char value) { return successive_capitalization_state{value}; });

  auto op = successive_capitalization_op{};

  auto d_result = scan_copy_if(  //
    input.begin(),
    input.end(),
    op,
    op);

  auto h_result = thrust::host_vector<successive_capitalization_state>(d_result.size());

  hipMemcpy(h_result.data(),  //
             d_result.data(),
             d_result.size() * sizeof(successive_capitalization_state),
             hipMemcpyDeviceToHost);

  ASSERT_EQ(static_cast<uint32_t>(3), h_result.size());

  EXPECT_EQ(static_cast<char>('G'), h_result[0].curr);
  EXPECT_EQ(static_cast<char>('L'), h_result[1].curr);
  EXPECT_EQ(static_cast<char>('K'), h_result[2].curr);
}

CUDF_TEST_PROGRAM_MAIN()
