#include <cudf_test/base_fixture.hpp>
#include <cudf_test/cudf_gtest.hpp>

#include "gtest/gtest.h"
#include "scan_select_if.cuh"

#include <cudf/utilities/span.hpp>

#include <rmm/thrust_rmm_allocator.h>

#include <thrust/iterator/constant_iterator.h>

class InclusiveCopyIfTest : public cudf::test::BaseFixture {
};

struct simple_op {
  inline __device__ uint32_t operator()(uint32_t lhs, uint32_t rhs) { return rhs + lhs; }
  inline __device__ bool operator()(uint32_t value)
  {
    printf("pred_op(%i)\n", value);
    return true;
  }
};

TEST_F(InclusiveCopyIfTest, CanScanSelectIf)
{
  auto input = thrust::make_constant_iterator<uint32_t>(1);

  auto op = simple_op{};

  auto d_result = scan_select_if(input, input + 256, op, op);

  auto h_result = thrust::host_vector<uint32_t>(d_result.size());

  hipMemcpy(h_result.data(), d_result.data(), d_result.size(), hipMemcpyDeviceToHost);

  for (auto value : h_result) { EXPECT_EQ(static_cast<uint32_t>(-1), value); }

  FAIL();
}

CUDF_TEST_PROGRAM_MAIN()
