#include <cudf_test/base_fixture.hpp>
#include <cudf_test/cudf_gtest.hpp>

#include "gtest/gtest.h"
#include "scan_select_if.cuh"

#include <cudf/utilities/span.hpp>

#include <rmm/thrust_rmm_allocator.h>

#include <thrust/iterator/constant_iterator.h>

class InclusiveCopyIfTest : public cudf::test::BaseFixture {
};

struct simple_op {
  inline __device__ uint32_t operator()(uint32_t lhs, uint32_t rhs)
  {
    // printf("b(%i) t(%i) %i + %i\n", blockIdx.x, threadIdx.x, lhs, rhs);
    return lhs + rhs;
  }
  inline __device__ bool operator()(uint32_t value)
  {
    // printf("b(%i) t(%i) pred(%i)\n", blockIdx.x, threadIdx.x, value);
    return true;
  }
};

TEST_F(InclusiveCopyIfTest, CanScanSelectIf)
{
  auto input = thrust::make_constant_iterator<uint32_t>(1);

  auto op = simple_op{};

  const uint32_t size = 4096;

  thrust::host_vector<uint32_t> h_result = scan_select_if(input, input + size, op, op);

  // hipMemcpy(h_result.data(), d_result.data(), d_result.size(), hipMemcpyDeviceToHost);

  ASSERT_EQ(static_cast<uint32_t>(size), h_result.size());

  for (uint32_t i = 0; i < h_result.size(); i++) {
    ASSERT_EQ(static_cast<uint32_t>(i + 1), h_result[i]);
  }

  // FAIL();
}

CUDF_TEST_PROGRAM_MAIN()
