#include "hip/hip_runtime.h"
#include <cudf/io/detail/csv_record_offsets.cuh>
#include <cudf_test/base_fixture.hpp>
#include <limits>

class CsvStateMachineTest : public cudf::test::BaseFixture {
};

rmm::device_vector<char> to_device_vector(std::string input)
{
  return rmm::device_vector<char>(input.c_str(), input.c_str() + input.size());
}

template <typename T>
std::vector<uint64_t> to_host_vector(rmm::device_uvector<T> const& d_source, hipStream_t stream)
{
  auto h_result = std::vector<T>(d_source.size());

  hipMemcpyAsync(h_result.data(),  //
                  d_source.data(),
                  d_source.size() * sizeof(T),
                  hipMemcpyDeviceToHost,
                  stream);

  return h_result;
}

TEST_F(CsvStateMachineTest, CanDetectTerminatedRecords)
{
  hipStream_t stream = 0;
  auto d_input        = to_device_vector("single\ncolumn\ncsv\n");

  auto d_row_offsets = cudf::io::detail::csv_gather_row_offsets(d_input, {}, {}, stream);
  auto h_row_offsets = to_host_vector(d_row_offsets, stream);

  hipStreamSynchronize(stream);

  ASSERT_EQ(h_row_offsets.size(), static_cast<uint32_t>(3));

  EXPECT_EQ(static_cast<uint64_t>(0), h_row_offsets[0]);
  EXPECT_EQ(static_cast<uint64_t>(7), h_row_offsets[1]);
  EXPECT_EQ(static_cast<uint64_t>(14), h_row_offsets[2]);
}

TEST_F(CsvStateMachineTest, CanTransitionCsvStates)
{
  hipStream_t stream = 0;
  auto d_input        = to_device_vector(
    "hello, world\n"
    "and,\"not\nh,ing\"\n"
    "new\n"
    "hello, world\n"
    "and,\"not\nh,ing\"\n"
    "new\n"
    "hello, world\n"
    "and,\"not\nh,ing\"\n"
    "new\n");

  auto d_row_offsets = cudf::io::detail::csv_gather_row_offsets(d_input, {}, {}, stream);
  auto h_row_offsets = to_host_vector(d_row_offsets, stream);

  hipStreamSynchronize(stream);

  ASSERT_EQ(h_row_offsets.size(), static_cast<uint32_t>(9));

  EXPECT_EQ(static_cast<uint64_t>(0), h_row_offsets[0]);
  EXPECT_EQ(static_cast<uint64_t>(13), h_row_offsets[1]);
  EXPECT_EQ(static_cast<uint64_t>(29), h_row_offsets[2]);

  EXPECT_EQ(static_cast<uint64_t>(33), h_row_offsets[3]);
  EXPECT_EQ(static_cast<uint64_t>(46), h_row_offsets[4]);
  EXPECT_EQ(static_cast<uint64_t>(62), h_row_offsets[5]);

  EXPECT_EQ(static_cast<uint64_t>(66), h_row_offsets[6]);
  EXPECT_EQ(static_cast<uint64_t>(79), h_row_offsets[7]);
  EXPECT_EQ(static_cast<uint64_t>(95), h_row_offsets[8]);
}

TEST_F(CsvStateMachineTest, CanTransitionCsvStatesWithRowRange)
{
  hipStream_t stream = 0;
  auto d_input        = to_device_vector(
    "hello, world\n"
    "and,\"not\nh,ing\"\n"
    "new\n"
    "hello, world\n"
    "and,\"not\nh,ing\"\n"
    "new\n"
    "hello, world\n"
    "and,\"not\nh,ing\"\n"
    "new\n");

  auto d_row_offsets = cudf::io::detail::csv_gather_row_offsets(  //
    d_input,
    {},
    {
      0,
      std::numeric_limits<uint64_t>::max(),
      4,
      7,
    });

  auto h_row_offsets = to_host_vector(d_row_offsets, stream);

  hipStreamSynchronize(stream);

  ASSERT_EQ(h_row_offsets.size(), static_cast<uint32_t>(3));

  EXPECT_EQ(static_cast<uint64_t>(33), h_row_offsets[0]);
  EXPECT_EQ(static_cast<uint64_t>(46), h_row_offsets[1]);
  EXPECT_EQ(static_cast<uint64_t>(62), h_row_offsets[2]);
}

TEST_F(CsvStateMachineTest, CanTransitionCsvStatesWithByteRange)
{
  hipStream_t stream = 0;

  auto d_input = to_device_vector(
    "hello, world\n"
    "and,\"not\nh,ing\"\n"
    "new\n"
    "hello, world\n"
    "and,\"not\nh,ing\"\n"
    "new\n"
    "hello, world\n"
    "and,\"not\nh,ing\"\n"
    "new\n");

  auto d_row_offsets = cudf::io::detail::csv_gather_row_offsets(  //
    d_input,
    {},
    {33, 63});

  auto h_row_offsets = to_host_vector(d_row_offsets, stream);

  hipStreamSynchronize(stream);

  ASSERT_EQ(h_row_offsets.size(), static_cast<uint32_t>(3));

  EXPECT_EQ(static_cast<uint64_t>(33), h_row_offsets[0]);
  EXPECT_EQ(static_cast<uint64_t>(46), h_row_offsets[1]);
  EXPECT_EQ(static_cast<uint64_t>(62), h_row_offsets[2]);
}

TEST_F(CsvStateMachineTest, CanTransitionStateSegments)
{
  using namespace cudf::io::detail;

  EXPECT_EQ(csv_state::record_end, static_cast<csv_state>(csv_superstate()));
}

TEST_F(CsvStateMachineTest, CanTransitionCsvStates2)
{
  using namespace cudf::io::detail;

  auto a = csv_superstate() + csv_token::comment + csv_token::other;
  auto b = csv_superstate() + csv_token::newline + csv_token::other;

  auto result = a + b;

  EXPECT_EQ(csv_state::field, static_cast<csv_state>(result));
}

CUDF_TEST_PROGRAM_MAIN()
